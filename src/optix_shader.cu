#include "hip/hip_runtime.h"
#include <optix.h>

#include "optix_shader.hpp"
#include "optix_device.cuh"

using namespace RLpbr::optix;

extern "C" {
__constant__ ShaderParams params;
}

struct CameraRay {
    float3 origin;
    float3 direction;
};

__device__ __forceinline__ CameraRay computeCameraRay(
    const CameraParams &camera, uint3 idx, uint3 dim)
{
    float4 data0 = camera.data[0];
    float4 data1 = camera.data[1];
    float4 data2 = camera.data[2];

    float3 origin = make_float3(data0.x, data0.y, data0.z);
    float3 view = make_float3(data0.w, data1.x, data1.y);
    float3 up = make_float3(data1.z, data1.w, data2.x);
    float3 right = make_float3(data2.y, data2.z, data2.w);

    float2 screen = make_float2((2.f * idx.x + 1) / dim.x - 1,
                                (2.f * idx.y + 1) / dim.y - 1);

    float3 direction = right * screen.x + up * screen.y + view;

    return CameraRay {
        origin,
        direction,
    };
}

__device__ __forceinline__ float3 computeBarycentrics()
{
    float2 attrs  = optixGetTriangleBarycentrics();

    return make_float3(1.f - attrs.x - attrs.y, attrs.x, attrs.y);
}

extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    uint3 idx = optixGetLaunchIndex();
    uint3 dim = optixGetLaunchDimensions();

    uint batch_idx = idx.z;

    const CameraParams &cam = params.cameras[batch_idx];

    auto [ray_origin, ray_dir] = computeCameraRay(cam, idx, dim);

    // Trace the ray against our scene hierarchy
    unsigned int payload_0;
    optixTrace(
            params.accelStructs[batch_idx],
            ray_origin,
            ray_dir,
            0.0f,                // Min intersection distance
            1e16f,               // Max intersection distance
            0.0f,                // rayTime -- used for motion blur
            OptixVisibilityMask(255), // Specify always visible
            OPTIX_RAY_FLAG_NONE,
            0,                   // SBT offset   -- See SBT discussion
            1,                   // SBT stride   -- See SBT discussion
            0,                   // missSBTIndex -- See SBT discussion
            payload_0);

    float depth = int_as_float(payload_0);

    // Record results in our output raster
    params.outputBuffer[idx.z * dim.y * dim.x + idx.y * dim.x + idx.x] = depth;
}

extern "C" __global__ void __miss__ms()
{
    optixSetPayload_0(float_as_int(0.f));
}

extern "C" __global__ void __closesthit__ch()
{
    float3 scaled_dir = optixGetWorldRayDirection() * optixGetRayTmax();

    float depth = length(scaled_dir);

    optixSetPayload_0(float_as_int(depth));
}
