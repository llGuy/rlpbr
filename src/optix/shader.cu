#include "hip/hip_runtime.h"
#include "device.cuh"
#include "sampler.cuh"
#include "shader.hpp"

#include <optix.h>
#include <cuda/std/tuple>
#include <hip/hip_math_constants.h>

using namespace RLpbr::optix;
using namespace cuda::std;

extern "C" {
__constant__ LaunchInput launchInput;
}

namespace ShaderConstants {
    static half *outputBuffer = (half *)OUTPUT_PTR;
    static const OptixTraversableHandle *accelStructs =
        (OptixTraversableHandle *)ACCEL_PTR;
    static const CameraParams *cameras =
        (CameraParams *)CAMERA_PTR;
    static const ClosestHitEnv *envs =
        (ClosestHitEnv *)ENV_PTR;
};

struct DeviceVertex {
    float3 position;
    float3 normal;
    float2 uv;
};

struct Camera {
    float3 origin;
    float3 view;
    float3 up;
    float3 right;
};

struct Triangle {
    DeviceVertex a;
    DeviceVertex b;
    DeviceVertex c;
};

__forceinline__ Camera unpackCamera(const CameraParams &packed)
{
    float4 data0 = packed.data[0];
    float4 data1 = packed.data[1];
    float4 data2 = packed.data[2];

    float3 origin = make_float3(data0.x, data0.y, data0.z);
    float3 view = make_float3(data0.w, data1.x, data1.y);
    float3 up = make_float3(data1.z, data1.w, data2.x);
    float3 right = make_float3(data2.y, data2.z, data2.w);

    return Camera {
        origin,
        view,
        up,
        right,
    };
}

__forceinline__ ClosestHitEnv unpackEnv(const ClosestHitEnv &env)
{
    return env;
}

__forceinline__ pair<float3, float3> computeCameraRay(
    const Camera &camera, uint3 idx, uint3 dim, Sampler &sampler)
{
    float2 jittered_raster = make_float2(idx.x, idx.y) + sampler.get2D();

    float2 screen = make_float2((2.f * jittered_raster.x) / dim.x - 1,
                                (2.f * jittered_raster.y) / dim.y - 1);

    float3 direction = camera.right * screen.x + camera.up * screen.y +
        camera.view;

    return {
        camera.origin,
        direction,
    };
}

__forceinline__ float computeDepth()
{
    float3 scaled_dir = optixGetWorldRayDirection() * optixGetRayTmax();
    return length(scaled_dir);
}

__forceinline__ float3 computeBarycentrics(float2 raw)
{
    return make_float3(1.f - raw.x - raw.y, raw.x, raw.y);
}

__forceinline__ uint32_t unormFloat2To32(float2 a)
{
    auto conv = [](float v) { return (uint32_t)trunc(v * 65535.f + 0.5f); };

    return conv(a.x) << 16 | conv(a.y);
}

__forceinline__ float2 unormFloat2From32(uint32_t a)
{
     return make_float2(a >> 16, a & 0xffff) * (1.f / 65535.f);
}

__forceinline__ void setHitPayload(float2 barycentrics,
                                   uint triangle_index,
                                   OptixTraversableHandle inst_hdl)
{
    optixSetPayload_0(unormFloat2To32(barycentrics));
    optixSetPayload_1(triangle_index);
    optixSetPayload_2(inst_hdl >> 32);
    optixSetPayload_3(inst_hdl & 0xFFFFFFFF);
}

__forceinline__ void setOutput(half *base_output, float3 rgb)
{
    base_output[0] = __float2half(rgb.x);
    base_output[1] = __float2half(rgb.y);
    base_output[2] = __float2half(rgb.z);
}

__forceinline__ DeviceVertex unpackVertex(
    const PackedVertex &packed)
{
    float4 a = packed.data[0];
    float4 b = packed.data[1];

    return DeviceVertex {
        make_float3(a.x, a.y, a.z),
        make_float3(a.w, b.x, b.y),
        make_float2(b.z, b.w),
    };
}

__forceinline__ Triangle fetchTriangle(
    const PackedVertex *vertex_buffer,
    const uint32_t *index_start)
{
    return Triangle {
        unpackVertex(vertex_buffer[index_start[0]]),
        unpackVertex(vertex_buffer[index_start[1]]),
        unpackVertex(vertex_buffer[index_start[2]]),
    };
}

__forceinline__ DeviceVertex interpolateTriangle(
    const Triangle &tri, float3 barys)
{
    return DeviceVertex {
        tri.a.position * barys.x +
            tri.b.position * barys.y + 
            tri.c.position * barys.z,
        tri.a.normal * barys.x +
            tri.b.normal * barys.y +
            tri.c.normal * barys.z,
        tri.a.uv * barys.x +
            tri.b.uv * barys.y +
            tri.c.uv * barys.z,
    };
}

// Returns *unnormalized vector*
inline float3 computeGeometricNormal(const Triangle &tri)
{
    float3 v1 = tri.b.position - tri.a.position;
    float3 v2 = tri.c.position - tri.a.position;

    return cross(v1, v2);
}

__forceinline__ float3 transformNormal(const float4 *w2o, float3 n)
{
    float4 r1 = w2o[0];
    float4 r2 = w2o[1];
    float4 r3 = w2o[2];

    return make_float3(
        r1.x * n.x + r2.x * n.y + r3.x * n.z,
        r1.y * n.x + r2.y * n.y + r3.y * n.z,
        r1.z * n.x + r2.z * n.y + r3.z * n.z);

}

__forceinline__ float3 transformPosition(const float4 *o2w, float3 p)
{
    float4 r1 = o2w[0];
    float4 r2 = o2w[1];
    float4 r3 = o2w[2];

    return make_float3(
        r1.x * p.x + r1.y * p.y + r1.z * p.z + r1.w,
        r2.x * p.x + r2.y * p.y + r2.z * p.z + r2.w,
        r3.x * p.x + r3.y * p.y + r3.z * p.z + r3.w);
}

inline float3 faceforward(const float3& n, const float3& i, const float3& nref)
{
  return n * copysignf( 1.0f, dot(i, nref) );
}

// Ray Tracing Gems Chapter 6 (avoid self intersections)
inline float3 offsetRayOrigin(const float3 &o, const float3 &geo_normal)
{
    constexpr float global_origin = 1.f / 32.f;
    constexpr float float_scale = 1.f / 65536.f;
    constexpr float int_scale = 256.f;

    int3 int_offset = make_int3(
        geo_normal.x * int_scale, geo_normal.y * int_scale,
        geo_normal.z * int_scale);

    float3 o_integer = make_float3(
        __int_as_float(
            __float_as_int(o.x) + ((o.x < 0) ? -int_offset.x : int_offset.x)),
        __int_as_float(
            __float_as_int(o.y) + ((o.y < 0) ? -int_offset.y : int_offset.y)),
        __int_as_float(
            __float_as_int(o.z) + ((o.z < 0) ? -int_offset.z : int_offset.z)));

    return make_float3(
        fabsf(o.x) < global_origin ?
            o.x + float_scale * geo_normal.x : o_integer.x,
        fabsf(o.y) < global_origin ?
            o.y + float_scale * geo_normal.y : o_integer.y,
        fabsf(o.z) < global_origin ?
            o.z + float_scale * geo_normal.z : o_integer.z);
}

extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    uint3 idx = optixGetLaunchIndex();
    uint3 dim = optixGetLaunchDimensions();

    uint batch_idx = launchInput.baseBatchOffset + idx.z;

    size_t base_out_offset = 
        3 * (batch_idx * dim.y * dim.x + idx.y * dim.x + idx.x);

    const Camera cam = unpackCamera(ShaderConstants::cameras[batch_idx]);
    const ClosestHitEnv ch_env = unpackEnv(ShaderConstants::envs[batch_idx]);
    const OptixTraversableHandle tlas =
        ShaderConstants::accelStructs[batch_idx];
    
    float3 pixel_radiance = make_float3(0.f);

    const float intensity = 10.f;

#if SPP != (1u)
#pragma unroll 1
#endif
    for (int32_t sample_idx = 0; sample_idx < SPP; sample_idx++) {
        Sampler sampler(idx, sample_idx, launchInput.baseFrameCounter);

        float3 sample_radiance = make_float3(0.f);
        float path_prob = 1.f;

        float3 next_origin;
        float3 next_direction;

#if MAX_DEPTH != (1u)
#pragma unroll 1
#endif
        for (int32_t path_depth = 0; path_depth < MAX_DEPTH;
             path_depth++) {
            float3 shade_origin;
            float3 shade_dir;
            if (path_depth == 0) {
                tie(shade_origin, shade_dir) =
                    computeCameraRay(cam, idx, dim, sampler);
            } else {
                shade_origin = next_origin;
                shade_dir = next_direction;
            }

            // Trace shade ray
            unsigned int payload_0;
            unsigned int payload_1;
            unsigned int payload_2;

            // Need to overwrite the register so miss detection works
            unsigned int payload_3 = 0;

            // FIXME Min T for both shadow and this ray
            optixTrace(
                    tlas,
                    shade_origin,
                    shade_dir,
                    0.f, // Min intersection distance
                    1e16f,               // Max intersection distance
                    0.0f,                // rayTime -- used for motion blur
                    OptixVisibilityMask(0xff), // Specify always visible
                    OPTIX_RAY_FLAG_NONE,
                    0,                   // SBT offset   -- See SBT discussion
                    0,                   // SBT stride   -- See SBT discussion
                    0,                   // missSBTIndex -- See SBT discussion
                    payload_0,
                    payload_1,
                    payload_2,
                    payload_3);

            // Miss, hit env map
            if (payload_3 == 0) {
                sample_radiance += intensity * path_prob;
                break;
            }

            float2 raw_barys = unormFloat2From32(payload_0);
            uint index_offset = payload_1;
            OptixTraversableHandle inst_hdl = (OptixTraversableHandle)(
                (uint64_t)payload_2 << 32 | (uint64_t)payload_3);

            const float4 *o2w =
                optixGetInstanceTransformFromHandle(inst_hdl);
            const float4 *w2o =
                optixGetInstanceInverseTransformFromHandle(inst_hdl);

            float3 barys = computeBarycentrics(raw_barys);

            Triangle hit_tri = fetchTriangle(ch_env.vertexBuffer,
                                             ch_env.indexBuffer + index_offset);
            DeviceVertex interpolated = interpolateTriangle(hit_tri, barys);
            float3 obj_geo_normal = computeGeometricNormal(hit_tri);

            float3 world_position =
                transformPosition(o2w, interpolated.position);
            float3 world_normal =
                transformNormal(w2o, interpolated.normal);
            float3 world_geo_normal =
                transformNormal(w2o, obj_geo_normal);

            world_normal = faceforward(world_normal, -shade_dir, world_normal);
            world_normal = normalize(world_normal);

            world_geo_normal = normalize(world_geo_normal);

            float3 up = make_float3(0, 0, 1);
            float3 up_alt = make_float3(0, 1, 0);

            float3 binormal = cross(world_normal, up);
            if (length(binormal) < 1e-3f) {
                binormal = cross(world_normal, up_alt);
            }
            binormal = normalize(binormal);

            float3 tangent = normalize(cross(binormal, world_normal));

            auto randomDirection = [&sampler] (const float3 &tangent,
                                           const float3 &binormal,
                                           const float3 &normal) {
                float2 uv = sampler.get2D();
                const float r = sqrtf(uv.x);
                const float phi = 2.0f * (HIP_PI_F) * uv.y;
                float2 disk = r * make_float2(cosf(phi), sinf(phi));
                float3 hemisphere = make_float3(disk.x, disk.y,
                    sqrtf(fmaxf(0.0f, 1.0f - dot(disk, disk))));

                return hemisphere.x * tangent +
                    hemisphere.y * binormal +
                    hemisphere.z * normal;
            };

            float3 shadow_origin =
                offsetRayOrigin(world_position, world_geo_normal);
            float3 shadow_direction =
                normalize(randomDirection(tangent, binormal, world_normal));

            payload_0 = 1;
            optixTrace(
                    tlas,
                    shadow_origin,
                    shadow_direction,
                    0.f,                // Min intersection distance
                    1e16f,               // Max intersection distance
                    0.0f,                // rayTime -- used for motion blur
                    OptixVisibilityMask(0xff), // Specify always visible
                    OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT |
                        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT |
                        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    0,                   // SBT offset   -- See SBT discussion
                    0,                   // SBT stride   -- See SBT discussion
                    0,                   // missSBTIndex -- See SBT discussion
                    payload_0,
                    payload_1,
                    payload_2,
                    payload_3);

            if (payload_0 == 0) {
                // "Shade"
                sample_radiance += intensity * path_prob;
            }

            // Start setup for next bounce
            next_origin = shadow_origin;
            next_direction = randomDirection(tangent, binormal, world_normal);

            // FIXME definitely wrong (light intensity?)
            path_prob *=
                1.f / (HIP_PI_F) * fabsf(dot(next_direction, world_normal));
        }

        pixel_radiance += sample_radiance / SPP;
    }

    setOutput(ShaderConstants::outputBuffer + base_out_offset, pixel_radiance);
}

extern "C" __global__ void __miss__ms()
{
    optixSetPayload_0(0);
}

extern "C" __global__ void __closesthit__ch()
{
    uint32_t base_index = optixGetInstanceId() + 3 * optixGetPrimitiveIndex();
    setHitPayload(optixGetTriangleBarycentrics(), base_index,
                  optixGetTransformListHandle(0));
}
